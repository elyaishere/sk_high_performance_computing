#include "hip/hip_runtime.h"
#include <chrono> 

#include "fft_cpu.hpp"

using namespace chrono;


template <typename T>
ostream &operator<<(ostream &o, vector<T> v) {
    if (v.size() > 0)
        o << v[0];
    for (size_t i = 1; i < v.size(); i++)
        o << " " << v[i];
    return o << endl;
}

static __device__ __host__ inline f2complex Add(f2complex A, f2complex B) {
    f2complex C;
    C.x = A.x + B.x;
    C.y = A.y + B.y;
    return C;
}

static __device__ __host__ inline f2complex Inverse(f2complex A) {
    f2complex C;
    C.x = -A.x;
    C.y = -A.y;
    return C;
}

static __device__ __host__ inline f2complex Multiply(f2complex A, f2complex B) {
    f2complex C;
    C.x = A.x * B.x - A.y * B.y;
    C.y = A.y * B.x + A.x * B.y;
    return C;
}

__global__ void inplace_divide_invert(f2complex *A, int n, int threads) {
    int i = blockIdx.x * threads + threadIdx.x;
    if (i < n) {
        A[i].x /= n;
        A[i].y /= n;
    }
}

__global__ void bitrev_reorder(f2complex *__restrict__ r, f2complex *__restrict__ d, int s, size_t nthr, int n) {
    int id = blockIdx.x * nthr + threadIdx.x;
    if (id < n and __brev(id) >> (32 - s) < n)
        r[__brev(id) >> (32 - s)] = d[id];
}


__device__ void inplace_fft_inner(f2complex *__restrict__ A, int i, int j, int len, int n, bool invert) {
    if (i + j + len / 2 < n and j < len / 2) {
        f2complex u, v;

        float angle = (2 * M_PI * j) / (len * (invert ? -1.0 : 1.0));
        v.x = cos(angle);
        v.y = sin(angle);

        u = A[i + j];
        v = Multiply(A[i + j + len / 2], v);
        A[i + j] = Add(u, v);
        A[i + j + len / 2] = Add(u, Inverse(v));
    }
}

__global__ void inplace_fft(f2complex *__restrict__ A, int i, int len, int n, int threads, bool invert) {
    int j = blockIdx.x * threads + threadIdx.x;
    inplace_fft_inner(A, i, j, len, n, invert);
}

__global__ void inplace_fft_outer(f2complex *__restrict__ A, int len, int n, int threads, bool invert)
{
    int i = blockIdx.x * threads + threadIdx.x;
    for (int j = 0; j < len / 2; j++) {
        inplace_fft_inner(A, i, j, len, n, invert);
    }
}

void cufft(vector<fcomplex> &a, bool invert, int balance = 10, int threads = 32) {

    int n = (int)a.size();
    int data_size = n * sizeof(f2complex);
    f2complex *data_array = (f2complex *)malloc(data_size);
    for (int i = 0; i < n; i++) {
        data_array[i].x = a[i].real();
        data_array[i].y = a[i].imag();
    }
    

    f2complex *A, *dn;
    hipMalloc((void **)&A, data_size);
    hipMalloc((void **)&dn, data_size);
    hipMemcpy(dn, data_array, data_size, hipMemcpyHostToDevice);

    int s = log2(n);

    bitrev_reorder<<<ceil(float(n) / threads), threads>>>(A, dn, s, threads, n);

    hipDeviceSynchronize();

    for (int len = 2; len <= n; len <<= 1) {
        if (n / len > balance) {
            inplace_fft_outer<<<ceil((float)n / threads), threads>>>(A, len, n, threads, invert);
        }
        else {
            for (int i = 0; i < n; i += len) {
                float repeats = len / 2;
                inplace_fft<<<ceil(repeats / threads), threads>>>(A, i, len, n, threads, invert);
            }
        }
    }
    
    if (invert)
        inplace_divide_invert<<<ceil(n * 1.00 / threads), threads>>>(A, n, threads);


    f2complex *result;
    result = (f2complex *)malloc(data_size);
    hipMemcpy(result, A, data_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
    {
        a[i] = fcomplex(result[i].x, result[i].y);
    }

    free(data_array);
    hipFree(A);
    hipFree(dn);
    return;
}

/// Function to multiply two polynomial with cuda
vector<int> cumult(vector<int> a, vector<int> b, int balance, int threads)
{
    vector<fcomplex> fa(a.begin(), a.end()), fb(b.begin(), b.end());

    // padding with zero to make their size equal to power of 2
    size_t n = 1;
    while (n < max(a.size(), b.size()))
        n <<= 1;
    n <<= 1;

    fa.resize(n), fb.resize(n);

    cufft(fa, false, balance, threads), cufft(fb, false, balance, threads);

    for (size_t i = 0; i < n; ++i)
        fa[i] *= fb[i];

    cufft(fa, true, balance, threads);

    vector<int> res;
    res.resize(n);
    for (size_t i = 0; i < n; ++i)
        res[i] = int(fa[i].real() + 0.5);

    return res;
}


#define N 1000
#define BALANCE 2

int main()
{
    std::vector<int> fa(N);
    std::generate(fa.begin(), fa.end(), std::rand);
    std::vector<int> fb(N);
    std::generate(fb.begin(), fb.end(), std::rand);
    freopen("out.txt", "w", stdout);
    auto multiplier = FFT();
    for(int threads = 1; threads <= 1024; threads++){

        auto start = high_resolution_clock::now(); 

        // cuda
        auto result_parallel = cumult(fa, fb, BALANCE, threads);

        auto stop = high_resolution_clock::now(); 
        auto duration = duration_cast<microseconds>(stop - start); 
      
        cout << threads << " " << duration.count() << " ";

        start = high_resolution_clock::now(); 
        // sequential
        auto result_sequential = multiplier.mult(fa, fb);

        stop = high_resolution_clock::now(); 
        duration = duration_cast<microseconds>(stop - start); 

        cout << duration.count() << endl;
       
    }
    
    
    return 0;
}
